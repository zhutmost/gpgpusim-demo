#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../include/kernel.cuh"

__global__ void kernel_add(int * a, int * b, int *c) {
    *c = *a + *b;
}

void my_add(int * a, int * b, int * c) {
    int * cu_a;
    int * cu_b;
    int * cu_c;
    hipMalloc((void **)&cu_a, sizeof(int));
    hipMalloc((void **)&cu_b, sizeof(int));
    hipMalloc((void **)&cu_c, sizeof(int));

    hipMemcpy(cu_a, a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cu_b, b, sizeof(int), hipMemcpyHostToDevice);
    kernel_add<<<1, 1>>>(cu_a, cu_b, cu_c);
    hipMemcpy(c, cu_c, sizeof(int), hipMemcpyDeviceToHost);

    return;
}
